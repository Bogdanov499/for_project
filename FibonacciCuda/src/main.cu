#include <iostream>
#include <vector>
#include <cstdint>
#include <hip/hip_runtime.h>

// Ядро: вычисление чисел Фибоначчи
__global__ void fibonacciKernel(uint64_t* d_fib, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i == 0) {
        d_fib[i] = 0;
    } else if (i == 1) {
        d_fib[i] = 1;
    } else if (i < N) {
        uint64_t a = 0, b = 1;
        for (int j = 2; j <= i; ++j) {
            uint64_t tmp = a + b;
            a = b;
            b = tmp;
        }
        d_fib[i] = b;
    }
}

// Хост-функция: вызов ядра и копирование данных
void computeFibonacciGPU(std::vector<uint64_t>& fib) {
    int N = fib.size();

    uint64_t* d_fib;
    hipMalloc(&d_fib, N * sizeof(uint64_t));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    fibonacciKernel<<<blocksPerGrid, threadsPerBlock>>>(d_fib, N);

    hipMemcpy(fib.data(), d_fib, N * sizeof(uint64_t), hipMemcpyDeviceToHost);

    hipFree(d_fib);
}

// Точка входа
int main() {
    const int N = 50;
    std::vector<uint64_t> fib(N);

    computeFibonacciGPU(fib);

    std::cout << "Первые " << N << " чисел Фибоначчи:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << fib[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}